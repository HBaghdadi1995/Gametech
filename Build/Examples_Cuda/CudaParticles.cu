#include "hip/hip_runtime.h"
#include "CudaParticles.cuh"
#include <random>


//This is run on the GPU, you can write (almost) c++ code and functions that
// run on the graphics card. These helper functions will get compiled both on the 
// CPU (__host__ tag) and GPU (__device__ tag) so can be accessed both inside
// kernals and in our program.
__host__ __device__ void initialize_y_vel(float3& vel)
{
	float vel_xy_sq = vel.x * vel.x + vel.z * vel.z;
	//Just a random func, to generate a nice(?) fountain effect for our particles
	vel.y = (7.5f - vel_xy_sq * vel_xy_sq) * 2.f;
	//vel.y = (15.f - vel_xy_sq); ///standard 'curve'
}




//This is run on the GPU, like an opengl shader except their is no "pipeline" only 
// code that runs per core (32 of these kernels per warp).
//
// The way each thread identifies itself is a little strange; the index value here
// is computed as (block_index * block_size + thread_index). As a reference, I have
// set the BlockDim for this kernal to 256, so threadIdx will go 0-255. The block
// dimension is an arbitary size of your choice (suggest power of 2), that corresponds
// to the amount of threads to run simultanously together. If your kernels require alot
// of memory (shared mem or local mem) or have set the L2 cache to be large, then you will
// be able to fit less threads into each block. If performance is critical, which if your
// using CUDA for your own projects, it probably is, then I suggest profiling the block
// dimension alot and attempting to reduce register usage as much as possible (see CUDA
// profiler for more details on kernal runtime)
__global__ void kernel_UpdateParticles(uint num_particles, float dt, float3* vertex_buf, float3* vel_arr) {
	///Note: This line will often look like an error because the defines are only valid with the cuda compiler
	///      not the c++ compiler. Just trust it /does/ compile fine :)
	uint index = blockIdx.x*blockDim.x + threadIdx.x; 
	if (index >= num_particles)
		return;

	//Read in memory for global memory
	float3 pos = vertex_buf[index];
	float3 vel = vel_arr[index];

	//Add gravity to particles
	vel.y -= 9.81f * dt;
	pos += vel * dt;


	//If pos goes below zero, lets just reset him at the centre of our 'emitter'
	if (pos.y < 0.0f)
	{
		pos = make_float3(0.f, 0.f, 0.f);

		initialize_y_vel(vel);
	}

	//Write our updated particle back to global memory
	vertex_buf[index] = pos;
	vel_arr[index] = vel;
}










//All the code below this point is ONLY executed on the CPU

CudaParticles::CudaParticles()
	: num_particles(0)
	, cGLPositions(NULL)
	, cArrVelocities(NULL)
{

}

CudaParticles::~CudaParticles()
{
	if (cArrVelocities)
	{
		gpuErrchk(hipFree(cArrVelocities));
		cArrVelocities = NULL;
	}

	if (cGLPositions)
	{
		gpuErrchk(hipGraphicsUnregisterResource(cGLPositions));
		cGLPositions = NULL;
	}
}


//For this example, we will be showing how easy it is to cross CUDA and opengl is,
// so instead of creating 2 cuda arrays, we will create a cuda array for velocities
// and use the pre-built opengl vertex buffer passed in (this could be from an nclgl
// mesh class etc)
void CudaParticles::InitializeArrays(uint size, GLuint glVertexBuffer)
{
	num_particles = size;

//Velocity Array - CUDA Array
	//First we need some 'random' velocities for our particles
	float3* tmp_vels = new float3[size];
	for (uint i = 0; i < size; ++i)
	{
		float angle = (float)DegToRad((rand() % 18000) / 50.f);
		float power = (rand() % 1000) / 500.f - 1.f;
		tmp_vels[i] = make_float3(
			cos(angle) * power,
			0.f,
			sin(angle)* power
		);
		


		initialize_y_vel(tmp_vels[i]);
	}

	//Allocate our GPU memory
	gpuErrchk(hipMalloc(&cArrVelocities, size * sizeof(float3)));

	//Copy our local memory over
	// - This is a generic copy function and as such we need to tell it if we
	//   are copying from the CPU to GPU or back again.
	gpuErrchk(hipMemcpy(cArrVelocities, tmp_vels, size * sizeof(float3), hipMemcpyHostToDevice));


	//Cleanup tmp velocities as they now happily live on the GPU
	delete[] tmp_vels;


//Position Array - OpenGL Buffer Resource
	//The map flags here can be none (read/write), writeDiscard (write only) or readOnly.
	gpuErrchk(hipGraphicsGLRegisterBuffer(&cGLPositions, glVertexBuffer, cudaGraphicsMapFlagsNone));
}


//We need some code to launch our kernel, this is executed on the CPU (__host__)
// and will split up our workload into 'blocks' and fire off all of the
// work for the GPU to split up and process using the above kernel code.
void CudaParticles::UpdateParticles(uint num_verts_to_update, float dt)
{
	num_verts_to_update = min(num_verts_to_update, num_particles);

	//First thing we need to do is 'lock' the opengl vertex buffer down, this
	// will lock it's memory in place and stop the driver from moving it's location
	// ..and also provide us with a memory pointer which we can use inside our kernel
	// to change read/write it's contents. :)
	size_t tmpVertexPtrSize;
	float3 *tmpVertexPtr;
	gpuErrchk(hipGraphicsMapResources(1, &cGLPositions, 0));
	gpuErrchk(hipGraphicsResourceGetMappedPointer((void **)&tmpVertexPtr, &tmpVertexPtrSize, cGLPositions));

	if (tmpVertexPtrSize < num_particles * sizeof(float3))
	{
		NCLERROR("OpenGL vertex buffer not large enough to encompass all our particles!");
		return;
	}


	// This is where we specify the block dimension, and number of blocks to fire
	// - See kernel_UpdateParticles comment for more information.
	// The y/z components here are always 1 as we only want to split along the x axis
	// for a single index value. For very large arrays, or 2D/3D textures etc you may
	// want index_x and index_y values aswell.
	dim3 block(256, 1, 1);
	/// We just want to do ceil(num_verts_to_update/block.x) here, but to avoid possible floating point
	/// errors, we can accomplish the same 'ceil' operation in integer math with: 
	/// [int->float->int]ceil(num_verts_to_update/block.x) = [integer only](num_verts_to_update + block.x - 1) / block.x
	dim3 grid((num_verts_to_update + block.x - 1) / block.x, 1, 1);

	//This is the code that actually 'runs' our kernel
	///This (again) will likely look like an intelisense error, but trust
	/// me, it /does/ compile alright.
	kernel_UpdateParticles<<< grid, block >>>(num_verts_to_update, dt, tmpVertexPtr, cArrVelocities);



	// Finally we need to unmap our OpenGL vertex buffer, allowing the driver
	// control over where it resides again.
	gpuErrchk(hipGraphicsUnmapResources(1, &cGLPositions, 0));
}
