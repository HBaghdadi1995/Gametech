#include "hip/hip_runtime.h"
#include "CudaCollidingParticles.cuh"


//When using the thrust library, anytime you want to use an anonomous function
// to process the array, you need to wrap it in a struct and pass that in instead.
//For example, this method is triggered by thrust for each element in our Particle
// array, and the output will is stored automatically in our openGL particle array.
struct CopyToOpenGL
{
	__host__ __device__
	float3 operator()(const Particle& p)
	{
		//Particles are go from 0 - grid width, and we want it to be centred on 0,0,0!
		const float world_dim = PARTICLE_GRID_SIZE * PARTICLE_GRID_CELL_SIZE;
		const float3 world_offset = make_float3(world_dim * 0.5f, 0.0f, world_dim * 0.5f);
		
		float3 centred_pos = p._pos - world_offset;
		return make_float3(centred_pos.x, centred_pos.y, centred_pos.z);
	}
};



/****************************
*** ALGORITHM EXPLANATION ***
*****************************/
//Parallel collision resolution:
//  - Making any serial algorithm parallel is very hard, and what
//    will almost certainly take up 99% of any GPU project. For this
//    example, collision resolution, we just take a n*2 approach.
//    Simply: For each collision, we process it twice, once for object A
//    and once for object B. The reason we do this is to avoid reading and 
//    writing to the same data at the same time (e.g. our physics constraints in parallel).
//    Instead, we allocate a thread to each particle, let it sum up all of the 'resolution'
//    forces acting on it from nearby collisions.
//    
//    On paper, this is just a much slower version of our CPU solver, though when split
//    onto hundreds of cores is still much faster than our CPU approach.

//How do we know which particles are neighbours?
//   - To do the collision resolution above, we need to know for each particle
//     which other particles are nearby and possibly colliding. To accomplish this
//     we do use a bucket sort. We generate a large 3D grid of cells and put each particle
//     into it's corresponding cell, resulting in finding all nearby particles a quick search
//     around the current and neighbouring grid cells and all their contained particles.
//
//If we have a fixed grid (like a texture) how do we place more than one particle in a single cell?
//   - Instead of having a static grid array, each grid cell just contains a start and end index which
//     points into the particle array. To generate this, we have to do a couple of steps:-
//		1: For each particle, compute it's grid cell index
//      2: Sort the particles by their grid cell indices
//      3. Run through the grid cell indices and save the 'start' of any grid cell change into our grid array
//      4. Run through the grid cell indices and save the 'end' of any grid cell change into our grid array
//

//-Footnote-
//       The result of this final codebase is actually very similar the CUDA "particles" example that comes
//       packaged with the samples. Their implementation is a bit faster, sorting lookups over entire particles
//       and using spring forces to resolve collisions in a more stable manner. If your interested, it's definetely
//       worth a look.
//
//       Another thing, for those that are interested, is a more descriptive explanation of how this works. It isn't
//       done exactly as mentioned in the article, as we don't create 8 seperate update kernels and instead just process
//       each collision pair twice. Though it explains the process much better, and is a more elegant solution to collision
//       resolution.
//		 https://developer.nvidia.com/gpugems/GPUGems3/gpugems3_ch32.html



 
__host__ __device__
int3 GetGridCell(const float3& pos)
{
	int3 cell;
	//Get a x,y,z cell index for the particle
	// Assumes positions go from 0 - (PARTICLE_GRID_SIZE * PARTICLE_GRID_CELL_SIZE)
	cell.x = static_cast<int>(pos.x / PARTICLE_GRID_CELL_SIZE);
	cell.y = static_cast<int>(pos.y / PARTICLE_GRID_CELL_SIZE);
	cell.z = static_cast<int>(pos.z / PARTICLE_GRID_CELL_SIZE);

	return cell;
}

__host__ __device__
uint GetGridCellHash(const int3& cell)
{
	//Generate a unique 'cell index' for the given cell.
	// - To handle 'edge' cases, we do a quick bitwise
	//   modulus to make sure all particles are correctly handled.

	int x = cell.x & (PARTICLE_GRID_SIZE - 1);
	int y = cell.y & (PARTICLE_GRID_SIZE - 1);
	int z = cell.z & (PARTICLE_GRID_SIZE - 1);

	return ((z * PARTICLE_GRID_SIZE) + x) * PARTICLE_GRID_SIZE + y;
}


//Bucket Sort: 1: For each particle, compute it's grid cell index
// Note: The other parts of the bucket sort list are all handled inside thrust library functions =]
struct GetCellGridIndex
{
	GetCellGridIndex() {}

	__host__ __device__
	uint operator()(const Particle& p) const
	{
		int3 cell = GetGridCell(p._pos);
		return GetGridCellHash(cell);
	}
};


//Given a particle p, check for and collide it with all particles in the given cell index
__device__
void CollideParticleWithCell(float baumgarte_factor, uint particle_idx, Particle& particle,	Particle& out_particle,
	int3 cell,
	Particle* all_particles, uint* grid_cell_start, uint* grid_cell_end)
{
	uint cellHash = GetGridCellHash(cell);

	//Get the start and end indices in the particle array which correspond
	// to the given grid cell
	uint arr_idx = grid_cell_start[cellHash];
	uint arr_end = grid_cell_end[cellHash];

	for (; arr_idx < arr_end; arr_idx++)
	{
		//Make sure we don't collide with ourselves!
		if (arr_idx == particle_idx)
			continue;

		Particle other_particle = all_particles[arr_idx];
		
		//Do a quick sphere-sphere test
		float3 ab = other_particle._pos - particle._pos;
		float lengthSq = dot(ab, ab);

		const float diameterSq = PARTICLE_RADIUS * PARTICLE_RADIUS * 4.f;
		if (lengthSq < diameterSq)
		{
			//We have a collision!
			float len = sqrtf(lengthSq);
			float3 abn = ab / len;

			//Direct normal collision (no friction/shear)
			float abnVel = dot(other_particle._vel - particle._vel, abn);		
			float jn = -(abnVel * (1.f + COLLISION_ELASTICITY));

			//Extra energy to overcome overlap error
			float overlap = PARTICLE_RADIUS * 2.f - len;
			float b = overlap * baumgarte_factor;
			
			//Normally we just add correctional energy (b) to our velocity,
			// but with such small particles and so many collisions this quickly gets 
			// out of control! The other way to solve positional errors is to move
			// the positions of the spheres, though this has numerous other problems and 
			// is ruins our collision neighbour checks. Though in general, velocity correction
			// adds energy and positional correction removes energy (and is unstable with the 
			// way we check collisions) so for now, we'll just use a half of each. Try messing
			// around with these values though! :)
			jn += b;
			//out_particle._pos -= abn * overlap * 0.5f; //Half positional correction, half because were only applying to A and not A + B
			
			
			jn = max(jn, 0.0f);
			//We just assume each particle is the same mass, so half the velocity change is applied to each.
			out_particle._vel -= abn * (jn * 0.5f);
		}

	}
}

__global__
void CollideParticles(float baumgarte_factor, uint num_particles, Particle* particles, Particle* out_particles, uint* grid_cell_start, uint* grid_cell_end)
{
	uint index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index >= num_particles)
		return;

	//For each particle, check for and collide it with all neighbouring particles.
	//  - As we know the particle radius is never larger than the grid cell size we only
	//    ever have to check in a one cell radius around (and including) our grid cell.
	Particle p = particles[index];
	Particle out_p = p;
	int3 cell = GetGridCell(p._pos);

	for (int z = -1; z <= 1; ++z)
	{
		for (int x = -1; x <= 1; ++x)
		{
			for (int y = -1; y <= 1; ++y)
			{
				int3 check_cell_idx = cell + make_int3(x, y, z);
				CollideParticleWithCell(baumgarte_factor, index, p, out_p, check_cell_idx, particles, grid_cell_start, grid_cell_end);

			}
		}
	}

	out_particles[index] = out_p;
}


// Update particle positions
// - Also handles boundary resolution. We don't want our particles
//   leaving our lookup grid.
struct UpdatePositions
{
	UpdatePositions(float dt, float3 gravity)
		: _dt(dt)
		, _gravity(gravity)
		, _gridMaxBounds(PARTICLE_GRID_SIZE * PARTICLE_GRID_CELL_SIZE - PARTICLE_RADIUS)
	{
	}

	float _dt;
	float3 _gravity;
	float _gridMaxBounds;

	__host__ __device__
		void operator()(Particle& p)
	{
		//Time integration
		p._vel += _gravity;
		p._vel *= 0.999f;

		p._pos += p._vel * _dt;



		//Out of Bounds Check
		// - Horrible branching mess... Hopefully your a better programmer than me. :(

		//X
		if (p._pos.x < PARTICLE_RADIUS)
		{
			p._pos.x = PARTICLE_RADIUS;
			p._vel.x = fabs(p._vel.x) * COLLISION_ELASTICITY;
		}
		if (p._pos.x > _gridMaxBounds)
		{
			p._pos.x = _gridMaxBounds;
			p._vel.x = -fabs(p._vel.x) * COLLISION_ELASTICITY;
		}

		//Y
		if (p._pos.y < PARTICLE_RADIUS)
		{
			p._pos.y = PARTICLE_RADIUS;
			p._vel.y = fabs(p._vel.x) * COLLISION_ELASTICITY;
		}
		if (p._pos.y > _gridMaxBounds)
		{
			p._pos.y = _gridMaxBounds;
			p._vel.y = -fabs(p._vel.x) * COLLISION_ELASTICITY;
		}

		//Z
		if (p._pos.z < PARTICLE_RADIUS)
		{
			p._pos.z = PARTICLE_RADIUS;
			p._vel.z = fabs(p._vel.x) * COLLISION_ELASTICITY;
		}
		if (p._pos.z > _gridMaxBounds)
		{
			p._pos.z = _gridMaxBounds;
			p._vel.z = -fabs(p._vel.x) * COLLISION_ELASTICITY;
		}

	}
};











//All the code below this point is ONLY executed on the CPU

CudaCollidingParticles::CudaCollidingParticles()
	: num_particles(0)
	, particles_ping(NULL)
	, cGLOutPositions(NULL)
{

}

CudaCollidingParticles::~CudaCollidingParticles()
{
	if (particles_ping)
	{
		gpuErrchk(hipFree(particles_ping));
		gpuErrchk(hipFree(particles_pong));
		gpuErrchk(hipFree(particles_grid_cell_index));
		gpuErrchk(hipFree(grid_cell_start));
		gpuErrchk(hipFree(grid_cell_end));

		particles_ping = NULL;
	}

	if (cGLOutPositions)
	{
		gpuErrchk(hipGraphicsUnregisterResource(cGLOutPositions));
		cGLOutPositions = NULL;
	}
}



void CudaCollidingParticles::InitializeParticleDam(int dam_width, int dam_height, int dam_depth)
{
///This function could have been a lot simpler, but I wanted nicely compacted dam... >.>
	uint num_even_rowed_particles = dam_width * dam_depth * dam_height / 2;
	num_particles = num_even_rowed_particles + (dam_width - 1) * (dam_depth - 1) * dam_height / 2;

	//Allocate Particle Arrays
	gpuErrchk(hipMalloc(&particles_pong, num_particles * sizeof(Particle)));
	gpuErrchk(hipMalloc(&particles_grid_cell_index, num_particles * sizeof(uint)));


	//Allocate our lookup grid
	const uint num_grid_cells = PARTICLE_GRID_SIZE*PARTICLE_GRID_SIZE*PARTICLE_GRID_SIZE;

	gpuErrchk(hipMalloc(&grid_cell_start, num_grid_cells * sizeof(uint)));
	gpuErrchk(hipMalloc(&grid_cell_end, num_grid_cells * sizeof(uint)));



	//Generate initial Particle data for our dam
	const float sqrt2 = sqrt(2.f);

	const float3 dam_size = make_float3(
			dam_width * PARTICLE_RADIUS * 2.f,
			dam_height * PARTICLE_RADIUS * (2.f + sqrt2) * 0.5f,
			dam_depth * PARTICLE_RADIUS * 2.f);

	const float world_dim = PARTICLE_GRID_SIZE * PARTICLE_GRID_CELL_SIZE - PARTICLE_RADIUS * 2.f;
	const float3 world_size = make_float3(world_dim, world_dim, world_dim);
	
	float3 start_offset = world_size * 0.5f - dam_size * 0.5f;
	start_offset.y = 0.0f;

	Particle* tmp_particles = new Particle[num_particles];

	//Initialize all the even rows of the dam
	for (int y = 0; y < dam_height / 2; y++)
	{
		for (int z = 0; z < dam_depth; ++z)
		{
			for (int x = 0; x < dam_width; ++x)
			{
				Particle p;
				p._vel = make_float3(0.f, 0.f, 0.f);

				p._pos = PARTICLE_RADIUS * make_float3(
					1.0f + x * 2.f,
					1.0f + y * (2.f + sqrt2),
					1.0f + z * 2.f
				);
				p._pos += start_offset;

				int idx = ((y * dam_depth) + z) * dam_width + x;
				tmp_particles[idx] = p;
			}	
		}
	}

	//Initialize all the odd rows of the dam
	for (int y = 0; y < dam_height / 2; y++)
	{
		for (int z = 0; z < dam_depth - 1; ++z)
		{
			for (int x = 0; x < dam_width - 1; ++x)
			{
				Particle p;
				p._vel = make_float3(0.f, 0.f, 0.f);

				p._pos = PARTICLE_RADIUS * make_float3(
					2.f + x * 2.f,
					(1.f + sqrt2) + y * (2.f + sqrt2),
					2.f + z * 2.f
				);
				p._pos += start_offset;

				int idx = ((y * (dam_depth-1)) + z) * (dam_width-1) + x;
				tmp_particles[num_even_rowed_particles + idx] = p;
			}
		}
	}

	gpuErrchk(hipMalloc(&particles_ping, num_particles * sizeof(Particle)));
	gpuErrchk(hipMemcpy(particles_ping, tmp_particles, num_particles * sizeof(Particle), hipMemcpyHostToDevice));

	delete[] tmp_particles;
}

void CudaCollidingParticles::InitializeOpenGLVertexBuffer(GLuint buffer_idx)
{
	//As the number of particles in this example is generated by the above function, the
	// opengl array has to be allocated after and initialized here later.
	gpuErrchk(hipGraphicsGLRegisterBuffer(&cGLOutPositions, buffer_idx, cudaGraphicsMapFlagsNone));
}

void CudaCollidingParticles::UpdateParticles(float dt)
{
	//See "ALGORITHM EXPLANATION" (top of this file) for info on what is meant to be happening here.

	//Note: Gravity here is tiny! The reason being that of stability, as the particles themselves are
	// small, and the timestep is comparitively massive, we need to make sure the maximum movement
	// of each particle per timestep is small. Try messing around with it, it's also important
	// for our CPU physics engine aswell (but hopefully never been noticed ^^ ).
	// For stability, particle systems normally use spring based collision resolution instead which
	// handles correctional energy (our baumgarte scalar) more leanently.
	const float3 gravity = make_float3(0, -0.02f, 0);
	const uint num_grid_cells = PARTICLE_GRID_SIZE*PARTICLE_GRID_SIZE*PARTICLE_GRID_SIZE;
	const float fixed_timestep = 1.0f / 60.0f;
	




	//Integrate our particles through time
	// - thrust::for_each applies a given function to each element in the array
	thrust::for_each(
		thrust::device_ptr<Particle>(particles_ping),
		thrust::device_ptr<Particle>(particles_ping + num_particles),
		UpdatePositions(fixed_timestep, gravity));

	//Generate our grid cell indices
	// - thrust::transform calls a given function on each element in the first array
	//   and outputs the result into the second array.
	thrust::transform(
		thrust::device_ptr<Particle>(particles_ping),
		thrust::device_ptr<Particle>(particles_ping + num_particles),
		thrust::device_ptr<uint>(particles_grid_cell_index),
		GetCellGridIndex());


	//Sort our Particles based on their grid cell indices
	// - thrust::sort_by_key sorts both keys and values based on the key array passed in.
	// Note: Sorting is still very slow (comparitively) on the GPU and is one case where the
	//       CPU is still often faster. However, copying all our data back to the host, sorting
	//       and copying back to the device is not a feasible option. Though it's something
	//       to keep in mind when doing your own algorithms.
	thrust::sort_by_key(
		thrust::device_ptr<uint>(particles_grid_cell_index),
		thrust::device_ptr<uint>(particles_grid_cell_index + num_particles),
		thrust::device_ptr<Particle>(particles_ping));

	//Compute grid cell start indices
	// - Runs through the list of particle grid cell indices, and saves for each
	//   grid cell the point in the array where it first appears.
	thrust::counting_iterator<uint> search_begin(0u);
	thrust::lower_bound(
		thrust::device_ptr<uint>(particles_grid_cell_index),
		thrust::device_ptr<uint>(particles_grid_cell_index + num_particles),
		search_begin,
		search_begin + num_grid_cells,
		thrust::device_ptr<uint>(grid_cell_start));

	//Compute grid cell end indices
	// - Runs through the list of particle grid cell indices, and saves for each
	//   grid cell the point in the array where it last appears (+1).
	thrust::upper_bound(
		thrust::device_ptr<uint>(particles_grid_cell_index),
		thrust::device_ptr<uint>(particles_grid_cell_index + num_particles),
		search_begin,
		search_begin + num_grid_cells,
		thrust::device_ptr<uint>(grid_cell_end));


	//Handle our collision resolution
	// - For each particle, check and handle collisions with all neighbouring particles.
	// Thrust?? - To my knowledge, thrust doesn't allow you raw array access. Everything must be
	//            done with iterators - Which can be used for this function, but for me, was
	//            easier just to write our own kernel and just access the particle array directly.
	dim3 block(64, 1, 1);
	dim3 grid((num_particles + block.x - 1) / block.x, 1, 1);
	float baumgarte_factor = 0.05f / fixed_timestep;

	for (int i = 0; i < 10; ++i)
	{
		CollideParticles<<< grid, block >>>(baumgarte_factor, num_particles, particles_ping, particles_pong, grid_cell_start, grid_cell_end);
		std::swap(particles_ping, particles_pong);
		
		//Should really do boundary check's here...
	}


	//Finally, copy our particle positions to openGL to be renderered as particles.
	size_t tmpVertexPtrSize;
	float3 *tmpVertexPtr;
	gpuErrchk(hipGraphicsMapResources(1, &cGLOutPositions, 0));
	gpuErrchk(hipGraphicsResourceGetMappedPointer((void **)&tmpVertexPtr, &tmpVertexPtrSize, cGLOutPositions));

	if (tmpVertexPtrSize < num_particles * sizeof(float3))
	{
		NCLERROR("OpenGL vertex buffer not large enough to encompass all our particles!");
		return;
	}

	thrust::transform(
		thrust::device_ptr<Particle>(particles_ping),
		thrust::device_ptr<Particle>(particles_ping + num_particles),
		thrust::device_ptr<float3>(tmpVertexPtr),
		CopyToOpenGL());

	gpuErrchk(hipGraphicsUnmapResources(1, &cGLOutPositions, 0));
}